#include "hip/hip_runtime.h"
#include "../device.hpp"

#include <thrust/device_ptr.h>
#include <thrust/scan.h>

#include <stdio.h>
#include <iostream>

#define FULL_MASK 0xffffffff

namespace kfusion {
    namespace device {
// texture<int, 1, hipReadModeElementType> edgeTex;
        texture<int, 1, hipReadModeElementType> triTex;
        texture<int, 1, hipReadModeElementType> numVertsTex;
    }  // namespace device
}  // namespace kfusion

void kfusion::device::bindTextures(const int* /*edgeBuf*/, const int* triBuf, const int* numVertsBuf) {
    hipChannelFormatDesc desc = hipCreateChannelDesc<int>();
    // cudaSafeCall(hipBindTexture(0, edgeTex, edgeBuf, desc) );
    cudaSafeCall(hipBindTexture(0, triTex, triBuf, desc));
    cudaSafeCall(hipBindTexture(0, numVertsTex, numVertsBuf, desc));
}
void kfusion::device::unbindTextures() {
    // cudaSafeCall( hipUnbindTexture(edgeTex) );
    cudaSafeCall(hipUnbindTexture(numVertsTex));
    cudaSafeCall(hipUnbindTexture(triTex));
}

namespace kfusion {
    namespace device {
        __device__ int global_count = 0;
        __device__ int output_count;
        __device__ unsigned int blocks_done = 0;

        __kf_device__ void kfusion::device::CubeIndexEstimator::readTsdf(int x, int y, int z, float& f, float& weight) const {
            half2 aux = *volume(x, y, z);
            f = unpack_tsdf(aux, weight);
        }

        __kf_device__ int kfusion::device::CubeIndexEstimator::computeCubeIndex(int x, int y, int z, float f[8]) const {
            float weight;
            readTsdf(x, y, z, f[0], weight);
            if (weight < MIN_TSDF_WEIGHT)
                return 0;
            readTsdf(x + 1, y, z, f[1], weight);
            if (weight < MIN_TSDF_WEIGHT)
                return 0;
            readTsdf(x + 1, y + 1, z, f[2], weight);
            if (weight < MIN_TSDF_WEIGHT)
                return 0;
            readTsdf(x, y + 1, z, f[3], weight);
            if (weight < MIN_TSDF_WEIGHT)
                return 0;
            readTsdf(x, y, z + 1, f[4], weight);
            if (weight < MIN_TSDF_WEIGHT)
                return 0;
            readTsdf(x + 1, y, z + 1, f[5], weight);
            if (weight < MIN_TSDF_WEIGHT)
                return 0;
            readTsdf(x + 1, y + 1, z + 1, f[6], weight);
            if (weight < MIN_TSDF_WEIGHT)
                return 0;
            readTsdf(x, y + 1, z + 1, f[7], weight);
            if (weight < MIN_TSDF_WEIGHT)
                return 0;

            // calculate flag indicating if each vertex is inside or outside isosurface
            int cubeindex = 0;
            cubeindex     = int(f[0] < isoValue);
            cubeindex += int(f[1] < isoValue) * 2;
            cubeindex += int(f[2] < isoValue) * 4;
            cubeindex += int(f[3] < isoValue) * 8;
            cubeindex += int(f[4] < isoValue) * 16;
            cubeindex += int(f[5] < isoValue) * 32;
            cubeindex += int(f[6] < isoValue) * 64;
            cubeindex += int(f[7] < isoValue) * 128;

            return cubeindex;
        }

        __kf_device__ void kfusion::device::OccupiedVoxels::operator()() const {
            int x = threadIdx.x + blockIdx.x * CTA_SIZE_X;
            int y = threadIdx.y + blockIdx.y * CTA_SIZE_Y;

            if (__all_sync(FULL_MASK, x >= volume.dims.x) || __all_sync(FULL_MASK, y >= volume.dims.y)) {
                return;
            }

            int ftid    = Block::flattenedThreadId();
            int warp_id = Warp::id();
            int lane_id = Warp::laneId();

            volatile __shared__ int warps_buffer[WARPS_COUNT];

            for (int z = 0; z < volume.dims.z - 1; z++) {
                int numVerts = 0;
                ;
                if (x + 1 < volume.dims.x && y + 1 < volume.dims.y) {
                    float field[8];
                    int cubeindex = computeCubeIndex(x, y, z, field);

                    // read number of vertices from texture
                    numVerts = (cubeindex == 0 || cubeindex == 255) ? 0 : tex1Dfetch(numVertsTex, cubeindex);
                }
// A stub for Pascal/Maxwell
#if __HIPCC__ < 700
__syncthreads();
#endif
                int total = __popc(__ballot_sync(FULL_MASK, numVerts > 0));

                if (total == 0)
                    continue;

                if (lane_id == 0) {
                    int old               = atomicAdd(&global_count, total);
                    warps_buffer[warp_id] = old;
                }
                int old_global_voxels_count = warps_buffer[warp_id];

                int offs = Warp::binaryExclScan(__ballot_sync(FULL_MASK, numVerts > 0));

                if (old_global_voxels_count + offs < max_size && numVerts > 0) {
                    voxels_indices[old_global_voxels_count + offs]  = volume.dims.y * volume.dims.x * z + volume.dims.x * y + x;
                    vertices_number[old_global_voxels_count + offs] = numVerts;
                }

                bool full = old_global_voxels_count + total >= max_size;

                if (full)
                    break;

            } /* for(int z = 0; z < 128 - 1; z++) */

            /////////////////////////
            // prepare for future scans
            if (ftid == 0) {
                unsigned int total_blocks = gridDim.x * gridDim.y * gridDim.z;
                unsigned int value        = atomicInc(&blocks_done, total_blocks);

                // last block
                if (value == total_blocks - 1) {
                    output_count = min(max_size, global_count);
                    blocks_done  = 0;
                    global_count = 0;
                }
            }
        } /* operator () */

        __global__ void getOccupiedVoxelsKernel(const OccupiedVoxels ov) { ov(); }

        int getOccupiedVoxels(const TsdfVolume& volume, DeviceArray2D<int>& occupied_voxels) {
            OccupiedVoxels ov(volume);

            ov.voxels_indices  = occupied_voxels.ptr(0);
            ov.vertices_number = occupied_voxels.ptr(1);
            ov.max_size        = occupied_voxels.cols();

            dim3 block(OccupiedVoxels::CTA_SIZE_X, OccupiedVoxels::CTA_SIZE_Y);
            dim3 grid(divUp(volume.dims.x, block.x), divUp(volume.dims.y, block.y));

            getOccupiedVoxelsKernel<<<grid, block>>>(ov);
            cudaSafeCall(hipGetLastError());
            cudaSafeCall(hipDeviceSynchronize());

            int size;
            cudaSafeCall(hipMemcpyFromSymbol(&size, HIP_SYMBOL(output_count), sizeof(size)));
            return size;
        }

        int computeOffsetsAndTotalVertices(DeviceArray2D<int>& occupied_voxels) {

            //printf("[MC] Starting computeOffsetsAndTotalVertices\n");
            thrust::device_ptr<int> beg(occupied_voxels.ptr(1));
            thrust::device_ptr<int> end = beg + occupied_voxels.cols();

            thrust::device_ptr<int> out(occupied_voxels.ptr(2));
            //printf("[MC] Starting exclusive_scan\n");
            try {
                thrust::exclusive_scan(beg, end, out);
            }
            catch(thrust::system_error &e)
            {
                // output an error message and exit
                std::cout << "Error in thrust: " << e.what() << std::endl;
            }

            int lastElement, lastScanElement;

            DeviceArray<int> last_elem(occupied_voxels.ptr(1) + occupied_voxels.cols() - 1, 1);
            DeviceArray<int> last_scan(occupied_voxels.ptr(2) + occupied_voxels.cols() - 1, 1);

            //printf("[MC] Starting last_elem.download\n");

            last_elem.download(&lastElement);
            last_scan.download(&lastScanElement);

            return lastElement + lastScanElement;
        }

        __kf_device__ float3 kfusion::device::TrianglesGenerator::get_node_coo(int x, int y, int z) const {
            float3 coo = make_float3(x, y, z);
            coo += 0.5f;  // shift to volume cell center;

            coo.x *= cell_size.x;
            coo.y *= cell_size.y;
            coo.z *= cell_size.z;

            return coo;
        }

        __kf_device__ float3 kfusion::device::TrianglesGenerator::vertex_interp(float3 p0, float3 p1, float f0,
        float f1) const {
        float t = (isoValue - f0) / (f1 - f0 + 1e-15f);
        float x = p0.x + t * (p1.x - p0.x);
        float y = p0.y + t * (p1.y - p0.y);
        float z = p0.z + t * (p1.z - p0.z);
        return make_float3(x, y, z);
    }

    __kf_device__ int kfusion::device::TrianglesGenerator::vertex_id(int vx, int vy, int vz, int edge_index) const
    {
        const int orientation[12] = {0, 1, 0, 1, 0, 1, 0, 1, 2, 2, 2, 2};
        const int coord_offset[12*3] = {0, 0, 0, 1, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 1, 1, 0, 1, 0, 1, 1, 0,
                                             0, 1, 0, 0, 0, 1, 0, 0, 1, 1, 0, 0, 1, 0};
        const int *curr_off_ptr = coord_offset+edge_index*3;
        int flat = ((vx+curr_off_ptr[0])*volume_yzdim+(vy+curr_off_ptr[1])*(volume.dims.z+1)+(vz+curr_off_ptr[2]))*3+orientation[edge_index];
        return flat;

    }

    __kf_device__ void kfusion::device::TrianglesGenerator::operator()() const {
        int tid = threadIdx.x;
        int idx = (blockIdx.y * MAX_GRID_SIZE_X + blockIdx.x) * CTA_SIZE + tid;

        if (idx >= voxels_count)
            return;

        int voxel = occupied_voxels[idx];

        int z = voxel / (volume.dims.x * volume.dims.y);
        int y = (voxel - z * volume.dims.x * volume.dims.y) / volume.dims.x;
        int x = (voxel - z * volume.dims.x * volume.dims.y) - y * volume.dims.x;

        float f[8];
        int cubeindex = computeCubeIndex(x, y, z, f);

        /* calculate cell vertex positions */
        float3 v[8];
        v[0] = get_node_coo(x, y, z);
        v[1] = get_node_coo(x + 1, y, z);
        v[2] = get_node_coo(x + 1, y + 1, z);
        v[3] = get_node_coo(x, y + 1, z);
        v[4] = get_node_coo(x, y, z + 1);
        v[5] = get_node_coo(x + 1, y, z + 1);
        v[6] = get_node_coo(x + 1, y + 1, z + 1);
        v[7] = get_node_coo(x, y + 1, z + 1);

        /* find vertices where surface intersects the cube; use shared memory to avoid using local */
        __shared__ float3 vertlist[12][CTA_SIZE];

        vertlist[0][tid]  = vertex_interp(v[0], v[1], f[0], f[1]);
        vertlist[1][tid]  = vertex_interp(v[1], v[2], f[1], f[2]);
        vertlist[2][tid]  = vertex_interp(v[3], v[2], f[3], f[2]);
        vertlist[3][tid]  = vertex_interp(v[0], v[3], f[0], f[3]);
        vertlist[4][tid]  = vertex_interp(v[4], v[5], f[4], f[5]);
        vertlist[5][tid]  = vertex_interp(v[5], v[6], f[5], f[6]);
        vertlist[6][tid]  = vertex_interp(v[7], v[6], f[7], f[6]);
        vertlist[7][tid]  = vertex_interp(v[4], v[7], f[4], f[7]);
        vertlist[8][tid]  = vertex_interp(v[0], v[4], f[0], f[4]);
        vertlist[9][tid]  = vertex_interp(v[1], v[5], f[1], f[5]);
        vertlist[10][tid] = vertex_interp(v[2], v[6], f[2], f[6]);
        vertlist[11][tid] = vertex_interp(v[3], v[7], f[3], f[7]);
        __syncthreads();

        /* output triangle vertices and normals */
        int numVerts = tex1Dfetch(numVertsTex, cubeindex);

        for (int i = 0; i < numVerts; i += 3) {
            int index = vertex_ofssets[idx] + i;

            int v1 = tex1Dfetch(triTex, (cubeindex * 16) + i + 0);
            int v2 = tex1Dfetch(triTex, (cubeindex * 16) + i + 1);
            int v3 = tex1Dfetch(triTex, (cubeindex * 16) + i + 2);

            /* NOTE (dig15): the surface could be smoother if the normal weren't the same for each vertex of the triangle */
            float3 n = normalized(cross(vertlist[v3][tid] - vertlist[v1][tid], vertlist[v2][tid] - vertlist[v1][tid]));

            store_point(outputVertices, index + 0, vertlist[v1][tid]); //pose * vertlist[v1][tid], may need on norm too
            store_point(outputNormals, index + 0, n);
            outputVertexIndices[index + 0] = vertex_id(x,y,z,v1);

            store_point(outputVertices, index + 1, vertlist[v2][tid]); //pose *
            store_point(outputNormals, index + 1, n);
            outputVertexIndices[index + 1] = vertex_id(x,y,z,v2);

            store_point(outputVertices, index + 2, vertlist[v3][tid]); //pose *
            store_point(outputNormals, index + 2, n);
            outputVertexIndices[index + 2] = vertex_id(x,y,z,v3);


//            store_point(outputVertices, index + 0, v[0]);
//            store_point(outputNormals, index + 0, n);
//            store_point(outputVertices, index + 1, v[1]);
//            store_point(outputNormals, index + 1, n);
//            store_point(outputVertices, index + 2, v[2]);
//            store_point(outputNormals, index + 2, n);
        }
    }

    __kf_device__ void kfusion::device::TrianglesGenerator::store_point(float4* ptr, int index,
                                                                        const float3& vertex) const {
        ptr[index] = make_float4(vertex.x, vertex.y, vertex.z, 1.f);
    }

    __global__ void trianglesGeneratorKernel(const TrianglesGenerator tg) { tg(); }

    void generateTriangles(const TsdfVolume& volume, const DeviceArray2D<int>& occupied_voxels, const float3& volume_size,
                           const Aff3f& pose, DeviceArray<Point>& outputVertices,
                           DeviceArray<Normal>& outputNormals, DeviceArray<int>& outputVertexIndices) {
        int device;
        cudaSafeCall(hipGetDevice(&device));

        hipDeviceProp_t prop;
        cudaSafeCall(hipGetDeviceProperties(&prop, device));

        typedef TrianglesGenerator Tg;
        Tg tg(volume);

        tg.occupied_voxels = occupied_voxels.ptr(0);
        tg.vertex_ofssets  = occupied_voxels.ptr(2);
        tg.voxels_count    = occupied_voxels.cols();
        tg.cell_size.x     = volume_size.x / volume.dims.x;
        tg.cell_size.y     = volume_size.y / volume.dims.y;
        tg.cell_size.z     = volume_size.z / volume.dims.z;
        tg.outputVertices  = outputVertices;
        tg.outputNormals   = outputNormals;
        tg.outputVertexIndices = outputVertexIndices;

        tg.pose = pose;

        int block_size = 256;
        int blocks_num = divUp(tg.voxels_count, block_size);

        dim3 block(block_size);
        dim3 grid(min(blocks_num, Tg::MAX_GRID_SIZE_X), divUp(blocks_num, Tg::MAX_GRID_SIZE_X));

        trianglesGeneratorKernel<<<grid, block>>>(tg);
        cudaSafeCall(hipGetLastError());
        cudaSafeCall(hipDeviceSynchronize());
    }


}  // namespace device
}  // namespace kfusion
