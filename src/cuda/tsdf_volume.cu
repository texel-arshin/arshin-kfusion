#include "hip/hip_runtime.h"
#include "device.hpp"
#include "texture_binder.hpp"
#include "kfusion/cuda/warpfield.hpp"
#include <stdio.h>

#define FULL_MASK 0xffffffff

using namespace kfusion::device;

////////////////////////////////////////////////////////////////////////////////////////////////////////////////
/// Volume initialization

namespace kfusion
{
    namespace device
    {
        __global__ void clear_volume_kernel(TsdfVolume tsdf)
        {
            int x = threadIdx.x + blockIdx.x * blockDim.x;
            int y = threadIdx.y + blockIdx.y * blockDim.y;

            if (x < tsdf.dims.x && y < tsdf.dims.y)
            {
                half2 *beg = tsdf.beg(x, y);
                half2 *end = beg + tsdf.dims.x * tsdf.dims.y * tsdf.dims.z;

                for(half2* pos = beg; pos != end; pos = tsdf.zstep(pos))
                    *pos = pack_tsdf (0.f, 0.f);
            }
        }
    }
}

void kfusion::device::clear_volume(TsdfVolume volume)
{
    dim3 block (32, 8);
    dim3 grid (1, 1, 1);
    grid.x = divUp (volume.dims.x, block.x);
    grid.y = divUp (volume.dims.y, block.y);

    clear_volume_kernel<<<grid, block>>>(volume);
    cudaSafeCall ( hipGetLastError () );
    cudaSafeCall ( hipDeviceSynchronize() );
}

////////////////////////////////////////////////////////////////////////////////////////////////////////////////
/// Volume integration

namespace kfusion
{
    namespace device
    {
        texture<float, 2> dists_tex(0, hipFilterModePoint, hipAddressModeBorder, cudaCreateChannelDescHalf());

        struct TsdfIntegrator
        {
            Aff3f vol2cam;
            Projector proj;
            int2 dists_size;
            warp::WarpField wf;

            float tranc_dist_inv;

            TsdfIntegrator(warp::WarpField wf) : wf(wf) {}

            __kf_device__
            void operator()(TsdfVolume& volume) const
            {
                int x = blockIdx.x * blockDim.x + threadIdx.x;
                int y = blockIdx.y * blockDim.y + threadIdx.y;
                int z = blockIdx.z * blockDim.z + threadIdx.z;

                if (x >= volume.dims.x || y >= volume.dims.y || z >= volume.dims.z)
                    return;

//                if (x==0 && y==0) {
//                    printf("KERNEL ENTRY POINT");
//                }
//

                float3 vw;
                float3 zstep = make_float3(0.f, 0.f, volume.voxel_size.z);

//                float3 vc = make_float3(x * volume.voxel_size.x, y * volume.voxel_size.y, 0);
                float3 vc = make_float3((x + 0.5) * volume.voxel_size.x, (y + 0.5) * volume.voxel_size.y, (z + 0.5) * volume.voxel_size.z);
//                float3 zstep = make_float3(vol2cam.R.data[0].z, vol2cam.R.data[1].z, vol2cam.R.data[2].z) * volume.voxel_size.z;
//
//                float3 vx = make_float3(x * volume.voxel_size.x, y * volume.voxel_size.y, 0);
//                float3 vc = vol2cam * vx; //tranform from volume coo frame to camera one



                TsdfVolume::elem_type* vptr = volume(x, y, z);
                {
                    //vw = vc;
                    float4 wres;
                    wres = warp::warp_vertex_nograd(wf, vc);
                    vw = make_float3(wres.x, wres.y, wres.z);
                    vw = vol2cam * vw;
//                    if (x==243 && y==319) {
//                        printf("PATH: (%f, %f, %f)\n", vc.x, vc.y, vc.z);
//                    }
                    float2 coo = proj(vw);

//                    printf("COO: %f %f\n", coo.x, coo.y);

                    //#if defined __CUDA_ARCH__ && __CUDA_ARCH__ >= 300
                    // this is actually workaround for kepler. it doesn't return 0.f for texture
                    // fetches for out-of-border coordinates even for cudaaddressmodeborder mode
                    if (coo.x < 0 || coo.y < 0 || coo.x >= dists_size.x || coo.y >= dists_size.y)
                        return;
                    //#endif
                    float Dp = tex2D(dists_tex, coo.x, coo.y);
                    if(Dp == 0 || vw.z <= 0)
                        return;

                    float sdf = Dp - __fsqrt_rn(dot(vw, vw)); //Dp - norm(v)
                    //printf("sdf: %f\n", sdf);
                    //printf("Dp: %f\n", Dp);

                    if (sdf >= -volume.trunc_dist)
                    {
                        float tsdf = fmin(1.f, sdf * tranc_dist_inv);

                        //read and unpack
                        float weight_prev;
                        float tsdf_prev = unpack_tsdf (gmem::LdCs(vptr), weight_prev);

                        float tsdf_new = __fdividef(__fmaf_rn(tsdf_prev, weight_prev, tsdf*wres.w), weight_prev + wres.w);
                        float weight_new = min (weight_prev + wres.w, volume.max_weight);
                        //printf("tsdf_new: %f\n", tsdf_new);


                        //pack and write
                        gmem::StCs(pack_tsdf (tsdf_new, weight_new), vptr);
                        //printf("unpacked tsdf_new: %f\n", unpack_tsdf (gmem::LdCs(vptr)));
                    }
                }  // for(;;)
            }
        };

        __global__ void integrate_kernel( const TsdfIntegrator& integrator, TsdfVolume& volume) { integrator(volume); };





        struct TsdfIntegratorNoWarp
        {
            Aff3f vol2cam;
            Projector proj;
            int2 dists_size;

            float tranc_dist_inv;

            TsdfIntegratorNoWarp() {}

            __kf_device__
            void operator()(TsdfVolume& volume) const
            {
                int x = blockIdx.x * blockDim.x + threadIdx.x;
                int y = blockIdx.y * blockDim.y + threadIdx.y;

                if (x >= volume.dims.x || y >= volume.dims.y)
                    return;

//                if (x==0 && y==0) {
//                    printf("KERNEL ENTRY POINT");
//                }
//

                float3 vw;
                float3 zstep = make_float3(0.f, 0.f, volume.voxel_size.z);

//                float3 vc = make_float3(x * volume.voxel_size.x, y * volume.voxel_size.y, 0);
                float3 vc = make_float3((x + 0.5) * volume.voxel_size.x, (y + 0.5) * volume.voxel_size.y, 0.5*volume.voxel_size.z);
//                float3 zstep = make_float3(vol2cam.R.data[0].z, vol2cam.R.data[1].z, vol2cam.R.data[2].z) * volume.voxel_size.z;
//
//                float3 vx = make_float3(x * volume.voxel_size.x, y * volume.voxel_size.y, 0);
//                float3 vc = vol2cam * vx; //tranform from volume coo frame to camera one



                TsdfVolume::elem_type* vptr = volume.beg(x, y);
                for(int i = 0; i < volume.dims.z; ++i, vc += zstep, vptr = volume.zstep(vptr))
                {
                    vw = vc;
                    vw = vol2cam * vw;
//                    if (x==243 && y==319) {
//                        printf("PATH: (%f, %f, %f)\n", vc.x, vc.y, vc.z);
//                    }
                    float2 coo = proj(vw);

//                    printf("COO: %f %f\n", coo.x, coo.y);

                    //#if defined __CUDA_ARCH__ && __CUDA_ARCH__ >= 300
                    // this is actually workaround for kepler. it doesn't return 0.f for texture
                    // fetches for out-of-border coordinates even for cudaaddressmodeborder mode
                    if (coo.x < 0 || coo.y < 0 || coo.x >= dists_size.x || coo.y >= dists_size.y)
                        continue;
                    //#endif
                    float Dp = tex2D(dists_tex, coo.x, coo.y);
                    if(Dp == 0 || vw.z <= 0)
                        continue;

                    float sdf = Dp - __fsqrt_rn(dot(vw, vw)); //Dp - norm(v)
                    //printf("sdf: %f\n", sdf);
                    //printf("Dp: %f\n", Dp);

                    if (sdf >= -volume.trunc_dist)
                    {
                        float tsdf = fmin(1.f, sdf * tranc_dist_inv);

                        //read and unpack
                        float weight_prev;
                        float tsdf_prev = unpack_tsdf (gmem::LdCs(vptr), weight_prev);

                        float tsdf_new = __fdividef(__fmaf_rn(tsdf_prev, weight_prev, tsdf), weight_prev + 1);
                        float weight_new = min (weight_prev + 1, volume.max_weight);
                        //printf("tsdf_new: %f\n", tsdf_new);


                        //pack and write
                        gmem::StCs(pack_tsdf (tsdf_new, weight_new), vptr);
                        //printf("unpacked tsdf_new: %f\n", unpack_tsdf (gmem::LdCs(vptr)));
                    }
                }  // for(;;)
            }
        };

        __global__ void integrate_nowarp_kernel( const TsdfIntegratorNoWarp nwintegrator, TsdfVolume volume) { nwintegrator(volume); };
    }
}

void kfusion::device::integrate(const PtrStepSz<half>& dists, TsdfVolume& volume, const Aff3f& aff, const Projector& proj, warp::WarpField wf)
{
    TsdfIntegrator ti(wf);
    ti.dists_size = make_int2(dists.cols, dists.rows);
    ti.vol2cam = aff;
    ti.proj = proj;
    ti.tranc_dist_inv = 1.f/volume.trunc_dist;

    dists_tex.filterMode = hipFilterModePoint;
    dists_tex.addressMode[0] = hipAddressModeBorder;
    dists_tex.addressMode[1] = hipAddressModeBorder;
    dists_tex.addressMode[2] = hipAddressModeBorder;
    TextureBinder binder(dists, dists_tex, cudaCreateChannelDescHalf()); (void)binder;

    TsdfIntegrator* cuda_ti;
    TsdfVolume* cuda_volume;
    cudaSafeCall ( hipMalloc(&cuda_ti, sizeof(TsdfIntegrator)) );
    cudaSafeCall ( hipMemcpy(cuda_ti, &ti, sizeof(TsdfIntegrator), hipMemcpyHostToDevice) );
    cudaSafeCall ( hipMalloc(&cuda_volume, sizeof(TsdfVolume)) );
    cudaSafeCall ( hipMemcpy(cuda_volume, &volume, sizeof(TsdfVolume), hipMemcpyHostToDevice) );
    dim3 block(1, 1, 32);
    dim3 grid(divUp(volume.dims.x, block.x), divUp(volume.dims.y, block.y), divUp(volume.dims.z, block.z));
    integrate_kernel<<<grid, block>>>(*cuda_ti, *cuda_volume);
    cudaSafeCall ( hipGetLastError () );
    cudaSafeCall ( hipFree(cuda_ti) );
    cudaSafeCall ( hipFree(cuda_volume) );
    cudaSafeCall ( hipDeviceSynchronize() );
}

void kfusion::device::integrate_nowarp(const PtrStepSz<half>& dists, TsdfVolume& volume, const Aff3f& aff, const Projector& proj)
{
    TsdfIntegratorNoWarp ti;
    ti.dists_size = make_int2(dists.cols, dists.rows);
    ti.vol2cam = aff;
    ti.proj = proj;
    ti.tranc_dist_inv = 1.f/volume.trunc_dist;

    dists_tex.filterMode = hipFilterModePoint;
    dists_tex.addressMode[0] = hipAddressModeBorder;
    dists_tex.addressMode[1] = hipAddressModeBorder;
    dists_tex.addressMode[2] = hipAddressModeBorder;
    TextureBinder binder(dists, dists_tex, cudaCreateChannelDescHalf()); (void)binder;

    dim3 block(32, 8);
    dim3 grid(divUp(volume.dims.x, block.x), divUp(volume.dims.y, block.y));
    integrate_nowarp_kernel<<<grid, block>>>(ti, volume);
    cudaSafeCall ( hipGetLastError () );
    cudaSafeCall ( hipDeviceSynchronize() );
}

////////////////////////////////////////////////////////////////////////////////////////////////////////////////
/// Volume ray casting

namespace kfusion
{
    namespace device
    {
        __kf_device__ void intersect(float3 ray_org, float3 ray_dir, /*float3 box_min,*/ float3 box_max, float &tnear, float &tfar)
        {
            const float3 box_min = make_float3(0.f, 0.f, 0.f);

            // compute intersection of ray with all six bbox planes
            float3 invR = make_float3(1.f/ray_dir.x, 1.f/ray_dir.y, 1.f/ray_dir.z);
            float3 tbot = invR * (box_min - ray_org);
            float3 ttop = invR * (box_max - ray_org);

            // re-order intersections to find smallest and largest on each axis
            float3 tmin = make_float3(fminf(ttop.x, tbot.x), fminf(ttop.y, tbot.y), fminf(ttop.z, tbot.z));
            float3 tmax = make_float3(fmaxf(ttop.x, tbot.x), fmaxf(ttop.y, tbot.y), fmaxf(ttop.z, tbot.z));

            // find the largest tmin and the smallest tmax
            tnear = fmaxf(fmaxf(tmin.x, tmin.y), fmaxf(tmin.x, tmin.z));
            tfar  = fminf(fminf(tmax.x, tmax.y), fminf(tmax.x, tmax.z));
        }

        template<typename Vol>
        __kf_device__ float interpolate(const Vol& volume, const float3& p_voxels)
        {
            float3 cf = p_voxels;
            cf.x-=0.5;
            cf.y-=0.5;
            cf.z-=0.5;

            //rounding to negative infinity
            int3 g = make_int3(__float2int_rd (cf.x), __float2int_rd (cf.y), __float2int_rd (cf.z));

            if (g.x < 0 || g.x >= volume.dims.x - 1 || g.y < 0 || g.y >= volume.dims.y - 1 || g.z < 0 || g.z >= volume.dims.z - 1)
                return 0.f;

            float a = cf.x - g.x;
            float b = cf.y - g.y;
            float c = cf.z - g.z;

            float tsdf = 0.f;
            tsdf += unpack_tsdf(*volume(g.x + 0, g.y + 0, g.z + 0)) * (1 - a) * (1 - b) * (1 - c);
            tsdf += unpack_tsdf(*volume(g.x + 0, g.y + 0, g.z + 1)) * (1 - a) * (1 - b) *      c;
            tsdf += unpack_tsdf(*volume(g.x + 0, g.y + 1, g.z + 0)) * (1 - a) *      b  * (1 - c);
            tsdf += unpack_tsdf(*volume(g.x + 0, g.y + 1, g.z + 1)) * (1 - a) *      b  *      c;
            tsdf += unpack_tsdf(*volume(g.x + 1, g.y + 0, g.z + 0)) *      a  * (1 - b) * (1 - c);
            tsdf += unpack_tsdf(*volume(g.x + 1, g.y + 0, g.z + 1)) *      a  * (1 - b) *      c;
            tsdf += unpack_tsdf(*volume(g.x + 1, g.y + 1, g.z + 0)) *      a  *      b  * (1 - c);
            tsdf += unpack_tsdf(*volume(g.x + 1, g.y + 1, g.z + 1)) *      a  *      b  *      c;
            return tsdf;
        }

        template<typename Vol>
        __kf_device__ float4 interpolate_grad(const Vol& volume, const float3& p_voxels)
        {
            float3 cf = p_voxels;
            cf.x-=0.5;
            cf.y-=0.5;
            cf.z-=0.5;

            //rounding to negative infinity
            int3 g = make_int3(__float2int_rd (cf.x), __float2int_rd (cf.y), __float2int_rd (cf.z));

            if (g.x < 0 || g.x >= volume.dims.x - 1 || g.y < 0 || g.y >= volume.dims.y - 1 || g.z < 0 || g.z >= volume.dims.z - 1)
                return make_float4(0., 0., 0., 0.);

            float a = cf.x - g.x;
            float b = cf.y - g.y;
            float c = cf.z - g.z;

            float tsdf = 0.f;
            float3 tsdf_grad = make_float3(0.,0.,0.);

            tsdf += unpack_tsdf(*volume(g.x + 0, g.y + 0, g.z + 0)) * (1 - a) * (1 - b) * (1 - c);
            tsdf += unpack_tsdf(*volume(g.x + 0, g.y + 0, g.z + 1)) * (1 - a) * (1 - b) *      c;
            tsdf += unpack_tsdf(*volume(g.x + 0, g.y + 1, g.z + 0)) * (1 - a) *      b  * (1 - c);
            tsdf += unpack_tsdf(*volume(g.x + 0, g.y + 1, g.z + 1)) * (1 - a) *      b  *      c;
            tsdf += unpack_tsdf(*volume(g.x + 1, g.y + 0, g.z + 0)) *      a  * (1 - b) * (1 - c);
            tsdf += unpack_tsdf(*volume(g.x + 1, g.y + 0, g.z + 1)) *      a  * (1 - b) *      c;
            tsdf += unpack_tsdf(*volume(g.x + 1, g.y + 1, g.z + 0)) *      a  *      b  * (1 - c);
            tsdf += unpack_tsdf(*volume(g.x + 1, g.y + 1, g.z + 1)) *      a  *      b  *      c;

            tsdf_grad.x += unpack_tsdf(*volume(g.x + 0, g.y + 0, g.z + 0)) * (-1) * (1 - b) * (1 - c);
            tsdf_grad.x += unpack_tsdf(*volume(g.x + 0, g.y + 0, g.z + 1)) * (-1) * (1 - b) *      c;
            tsdf_grad.x += unpack_tsdf(*volume(g.x + 0, g.y + 1, g.z + 0)) * (-1) *      b  * (1 - c);
            tsdf_grad.x += unpack_tsdf(*volume(g.x + 0, g.y + 1, g.z + 1)) * (-1) *      b  *      c;
            tsdf_grad.x += unpack_tsdf(*volume(g.x + 1, g.y + 0, g.z + 0)) *      1  * (1 - b) * (1 - c);
            tsdf_grad.x += unpack_tsdf(*volume(g.x + 1, g.y + 0, g.z + 1)) *      1  * (1 - b) *      c;
            tsdf_grad.x += unpack_tsdf(*volume(g.x + 1, g.y + 1, g.z + 0)) *      1  *      b  * (1 - c);
            tsdf_grad.x += unpack_tsdf(*volume(g.x + 1, g.y + 1, g.z + 1)) *      1  *      b  *      c;

            tsdf_grad.y += unpack_tsdf(*volume(g.x + 0, g.y + 0, g.z + 0)) * (1 - a) * (-1) * (1 - c);
            tsdf_grad.y += unpack_tsdf(*volume(g.x + 0, g.y + 0, g.z + 1)) * (1 - a) * (-1) *      c;
            tsdf_grad.y += unpack_tsdf(*volume(g.x + 0, g.y + 1, g.z + 0)) * (1 - a) *      1  * (1 - c);
            tsdf_grad.y += unpack_tsdf(*volume(g.x + 0, g.y + 1, g.z + 1)) * (1 - a) *      1  *      c;
            tsdf_grad.y += unpack_tsdf(*volume(g.x + 1, g.y + 0, g.z + 0)) *      a  * (-1) * (1 - c);
            tsdf_grad.y += unpack_tsdf(*volume(g.x + 1, g.y + 0, g.z + 1)) *      a  * (-1) *      c;
            tsdf_grad.y += unpack_tsdf(*volume(g.x + 1, g.y + 1, g.z + 0)) *      a  *      1  * (1 - c);
            tsdf_grad.y += unpack_tsdf(*volume(g.x + 1, g.y + 1, g.z + 1)) *      a  *      1  *      c;

            tsdf_grad.z += unpack_tsdf(*volume(g.x + 0, g.y + 0, g.z + 0)) * (1 - a) * (1 - b) * (-1);
            tsdf_grad.z += unpack_tsdf(*volume(g.x + 0, g.y + 0, g.z + 1)) * (1 - a) * (1 - b) *      1;
            tsdf_grad.z += unpack_tsdf(*volume(g.x + 0, g.y + 1, g.z + 0)) * (1 - a) *      b  * (-1);
            tsdf_grad.z += unpack_tsdf(*volume(g.x + 0, g.y + 1, g.z + 1)) * (1 - a) *      b  *      1;
            tsdf_grad.z += unpack_tsdf(*volume(g.x + 1, g.y + 0, g.z + 0)) *      a  * (1 - b) * (-1);
            tsdf_grad.z += unpack_tsdf(*volume(g.x + 1, g.y + 0, g.z + 1)) *      a  * (1 - b) *      1;
            tsdf_grad.z += unpack_tsdf(*volume(g.x + 1, g.y + 1, g.z + 0)) *      a  *      b  * (-1);
            tsdf_grad.z += unpack_tsdf(*volume(g.x + 1, g.y + 1, g.z + 1)) *      a  *      b  *      1;
            return make_float4(tsdf_grad.x, tsdf_grad.y, tsdf_grad.z, tsdf);
        }

        struct TsdfQuerier
        {
            TsdfVolume volume;
            float3 voxel_size_inv;
            TsdfQuerier(const TsdfVolume& volume): volume(volume) {}

            __kf_device__
            bool is_voxel_occupied(const float3& p) const
            {
                //rounding to negative infinity
                int x = __float2int_rd (p.x);
                int y = __float2int_rd (p.y);
                int z = __float2int_rd (p.z);
                if (x < 0 || x >= volume.dims.x || y < 0 || y >= volume.dims.y || z < 0 || z >= volume.dims.z)
                    return false;
                float w;
                unpack_tsdf(*volume(x, y, z), w);
                return w >= MIN_TSDF_WEIGHT;
            }

            __kf_device__
            bool is_voxel_grad_capable(const float3& p) const
            {
                float3 cf = p;
                cf.x-=0.5;
                cf.y-=0.5;
                cf.z-=0.5;

                //rounding to negative infinity
                int3 g = make_int3(__float2int_rd (cf.x), __float2int_rd (cf.y), __float2int_rd (cf.z));

                if (g.x < 0 || g.x >= volume.dims.x - 1 || g.y < 0 || g.y >= volume.dims.y - 1 || g.z < 0 || g.z >= volume.dims.z - 1)
                    return false;
                float w;
                for (char i = 0; i<2; i++) for (char j=0; j<2; j++) for (char k=0; k<2; k++)
                {
                    unpack_tsdf(*volume(g.x + i, g.y + j, g.z + k), w);
                    if (w < MIN_TSDF_WEIGHT)
                        return false;
                }
                return true;
            }

            __kf_device__
            float fetch_tsdf(const float3& p) const
            {
                //rounding to negative infinity
                int x = __float2int_rd (p.x);
                int y = __float2int_rd (p.y);
                int z = __float2int_rd (p.z);
                if (x < 0 || x >= volume.dims.x || y < 0 || y >= volume.dims.y || z < 0 || z >= volume.dims.z)
                    return 0.f;
                float w;
                float f = unpack_tsdf(*volume(x, y, z), w);
                return (w < MIN_TSDF_WEIGHT) ? 0.f : f;
            }

            __kf_device__
            void operator()(const float *verts, float *tsdfs, float *tsdfs_grad, int verts_count) const
            {
                int off = blockIdx.x * blockDim.x + threadIdx.x;
                int off3 = 3*off;
                if (off >= verts_count)
                    return;
                float4 tsdf_with_grad = make_float4(0.,0.,0.,0.);
                float3 voxel_coords = make_float3(verts[off3],verts[off3+1],verts[off3+2])*voxel_size_inv;
                if (is_voxel_occupied(voxel_coords))
                {
                    if (is_voxel_grad_capable(voxel_coords))
                        tsdf_with_grad = interpolate_grad(volume, voxel_coords);
                    else
                        tsdf_with_grad.w = fetch_tsdf(voxel_coords);
                }
                tsdfs[off] = tsdf_with_grad.w;
                tsdfs_grad[off3] = tsdf_with_grad.x*voxel_size_inv.x;
                tsdfs_grad[off3+1] = tsdf_with_grad.y*voxel_size_inv.y;
                tsdfs_grad[off3+2] = tsdf_with_grad.z*voxel_size_inv.z;
            }
//            __kf_device__
//            void operator()(const float *verts, float *tsdfs, int verts_count) const
//            {
//                int off = blockIdx.x * blockDim.x + threadIdx.x;
//                int off3 = 3*off;
//                if (off >= verts_count)
//                    return;
//                float3 voxel_coords = make_float3(verts[off3],verts[off3+1],verts[off3+2])*voxel_size_inv;
//                tsdfs[off] = fetch_tsdf(voxel_coords);
//            }
        };

        __global__ void queryvolume_with_grad_kernel(const TsdfQuerier querier, const float *verts, float *tsdfs, float *tsdfs_grad, int verts_count)
        { querier(verts, tsdfs, tsdfs_grad, verts_count); };

        struct TsdfRaycaster
        {
            TsdfVolume volume;

            Aff3f aff;
            Mat3f Rinv;

            Vec3f volume_size;
            Reprojector reproj;
            float time_step;
            float3 gradient_delta;
            float3 voxel_size_inv;

            TsdfRaycaster(const TsdfVolume& volume, const Aff3f& aff, const Mat3f& Rinv, const Reprojector& _reproj);

            __kf_device__
            float fetch_tsdf(const float3& p) const
            {
                //rounding to nearest even
                int x = __float2int_rn (p.x * voxel_size_inv.x);
                int y = __float2int_rn (p.y * voxel_size_inv.y);
                int z = __float2int_rn (p.z * voxel_size_inv.z);
                float w;
                float f = unpack_tsdf(*volume(x, y, z), w);
                return (w < MIN_TSDF_WEIGHT) ? 0.f : f;
            }

            __kf_device__
            void operator()(PtrStepSz<ushort> depth, PtrStep<Normal> normals) const
            {
                int x = blockIdx.x * blockDim.x + threadIdx.x;
                int y = blockIdx.y * blockDim.y + threadIdx.y;

                if (x >= depth.cols || y >= depth.rows)
                    return;

                const float qnan = numeric_limits<float>::quiet_NaN();

                depth(y, x) = 0;
                normals(y, x) = make_float4(qnan, qnan, qnan, qnan);

                float3 ray_org = aff.t;
                float3 ray_dir = normalized( aff.R * reproj(x, y, 1.f) );

                // We do subtract voxel size to minimize checks after
                // Note: origin of volume coordinate is placeed
                // in the center of voxel (0,0,0), not in the corener of the voxel!
                float3 box_max = volume_size - volume.voxel_size;

                float tmin, tmax;
                intersect(ray_org, ray_dir, box_max, tmin, tmax);

                const float min_dist = 0.f;
                tmin = fmax(min_dist, tmin);
                if (tmin >= tmax)
                    return;

                tmax -= time_step;
                float3 vstep = ray_dir * time_step;
                float3 next = ray_org + ray_dir * tmin;

                float tsdf_next = fetch_tsdf(next);

                for (float tcurr = tmin; tcurr < tmax; tcurr += time_step)
                {
                    float tsdf_curr = tsdf_next;
                    float3     curr = next;
                    next += vstep;

                    tsdf_next = fetch_tsdf(next);
                    if (tsdf_curr < 0.f && tsdf_next > 0.f)
                        break;

                    if (tsdf_curr > 0.f && tsdf_next < 0.f)
                    {
                        float Ft   = interpolate(volume, curr * voxel_size_inv);
                        float Ftdt = interpolate(volume, next * voxel_size_inv);

                        float Ts = tcurr - __fdividef(time_step * Ft, Ftdt - Ft);

                        float3 vertex = ray_org + ray_dir * Ts;
                        float3 normal = compute_normal(vertex);

                        if (!isnan(normal.x * normal.y * normal.z))
                        {
                            normal = Rinv * normal;
                            vertex = Rinv * (vertex - aff.t);

                            normals(y, x) = make_float4(normal.x, normal.y, normal.z, 0);
                            depth(y, x) = static_cast<ushort>(vertex.z * 1000);
                        }
                        break;
                    }
                } /* for (;;) */
            }

            __kf_device__
            void operator()(PtrStepSz<Point> points, PtrStep<Normal> normals) const
            {
                int x = blockIdx.x * blockDim.x + threadIdx.x;
                int y = blockIdx.y * blockDim.y + threadIdx.y;

                if (x >= points.cols || y >= points.rows)
                    return;

                const float qnan = numeric_limits<float>::quiet_NaN();

                points(y, x) = normals(y, x) = make_float4(qnan, qnan, qnan, qnan);

                float3 ray_org = aff.t;
                float3 ray_dir = normalized( aff.R * reproj(x, y, 1.f) );

                // We do subtract voxel size to minimize checks after
                // Note: origin of volume coordinate is placeed
                // in the center of voxel (0,0,0), not in the corener of the voxel!
                float3 box_max = volume_size - volume.voxel_size;

                float tmin, tmax;
                intersect(ray_org, ray_dir, box_max, tmin, tmax);

                const float min_dist = 0.f;
                tmin = fmax(min_dist, tmin);
                if (tmin >= tmax)
                    return;

                tmax -= time_step;
                float3 vstep = ray_dir * time_step;
                float3 next = ray_org + ray_dir * tmin;

                float tsdf_next = fetch_tsdf(next);
                //printf("TSDF unpacked: %f", tsdf_next);
                for (float tcurr = tmin; tcurr < tmax; tcurr += time_step)
                {
                    float tsdf_curr = tsdf_next;
                    float3     curr = next;
                    next += vstep;

                    tsdf_next = fetch_tsdf(next);
                    if (tsdf_curr < 0.f && tsdf_next > 0.f)
                        break;

                    if (tsdf_curr > 0.f && tsdf_next < 0.f)
                    {
                        float Ft   = interpolate(volume, curr * voxel_size_inv);
                        float Ftdt = interpolate(volume, next * voxel_size_inv);

                        float Ts = tcurr - __fdividef(time_step * Ft, Ftdt - Ft);

                        float3 vertex = ray_org + ray_dir * Ts;
                        float3 normal = compute_normal(vertex);

                        if (!isnan(normal.x * normal.y * normal.z))
                        {
//                            normal = Rinv * normal;
//                            vertex = Rinv * (vertex - aff.t);

                            normals(y, x) = make_float4(normal.x, normal.y, normal.z, 0.f);
                            points(y, x) = make_float4(vertex.x, vertex.y, vertex.z, 0.f);
                        }
                        break;
                    }
                } /* for (;;) */
            }


            __kf_device__
            float3 compute_normal(const float3& p) const
            {
                float3 n;

                float Fx1 = interpolate(volume, make_float3(p.x + gradient_delta.x, p.y, p.z) * voxel_size_inv);
                float Fx2 = interpolate(volume, make_float3(p.x - gradient_delta.x, p.y, p.z) * voxel_size_inv);
                n.x = __fdividef(Fx1 - Fx2, gradient_delta.x);

                float Fy1 = interpolate(volume, make_float3(p.x, p.y + gradient_delta.y, p.z) * voxel_size_inv);
                float Fy2 = interpolate(volume, make_float3(p.x, p.y - gradient_delta.y, p.z) * voxel_size_inv);
                n.y = __fdividef(Fy1 - Fy2, gradient_delta.y);

                float Fz1 = interpolate(volume, make_float3(p.x, p.y, p.z + gradient_delta.z) * voxel_size_inv);
                float Fz2 = interpolate(volume, make_float3(p.x, p.y, p.z - gradient_delta.z) * voxel_size_inv);
                n.z = __fdividef(Fz1 - Fz2, gradient_delta.z);

                return normalized (n);
            }
        };

        inline TsdfRaycaster::TsdfRaycaster(const TsdfVolume& _volume, const Aff3f& _aff, const Mat3f& _Rinv, const Reprojector& _reproj)
            : volume(_volume), aff(_aff), Rinv(_Rinv), reproj(_reproj) {}

        __global__ void raycast_kernel(const TsdfRaycaster raycaster, PtrStepSz<ushort> depth, PtrStep<Normal> normals)
        { raycaster(depth, normals); };

        __global__ void raycast_kernel(const TsdfRaycaster raycaster, PtrStepSz<Point> points, PtrStep<Normal> normals)
        { raycaster(points, normals); };

    }
}

void kfusion::device::query_volume_with_grad(const TsdfVolume& volume, const DeviceArray<float> &verts, DeviceArray<float> &tsdfs, DeviceArray<float> &tsdfs_grad)
{
    TsdfQuerier vq(volume);
    vq.voxel_size_inv = 1.f/volume.voxel_size;
    dim3 block(256, 1);
    dim3 grid (divUp (verts.size(), block.x), 1);
    queryvolume_with_grad_kernel<<<grid, block>>>(vq, verts.ptr(), tsdfs.ptr(), tsdfs_grad.ptr(), tsdfs.size());
    cudaSafeCall (hipGetLastError ());
    cudaSafeCall ( hipDeviceSynchronize() );
}

void kfusion::device::raycast(const TsdfVolume& volume, const Aff3f& aff, const Mat3f& Rinv, const Reprojector& reproj,
                              Depth& depth, Normals& normals, float raycaster_step_factor, float gradient_delta_factor)
{
    TsdfRaycaster rc(volume, aff, Rinv, reproj);

    rc.volume_size = volume.voxel_size * volume.dims;
    rc.time_step = volume.trunc_dist * raycaster_step_factor;
    rc.gradient_delta = volume.voxel_size * gradient_delta_factor;
    rc.voxel_size_inv = 1.f/volume.voxel_size;

    dim3 block(32, 8);
    dim3 grid (divUp (depth.cols(), block.x), divUp (depth.rows(), block.y));

    raycast_kernel<<<grid, block>>>(rc, (PtrStepSz<ushort>)depth, normals);
    cudaSafeCall (hipGetLastError ());
}


void kfusion::device::raycast(const TsdfVolume& volume, const Aff3f& aff, const Mat3f& Rinv, const Reprojector& reproj,
                              Points& points, Normals& normals, float raycaster_step_factor, float gradient_delta_factor)
{
    TsdfRaycaster rc(volume, aff, Rinv, reproj);

    rc.volume_size = volume.voxel_size * volume.dims;
    rc.time_step = volume.trunc_dist * raycaster_step_factor;
    rc.gradient_delta = volume.voxel_size * gradient_delta_factor;
    rc.voxel_size_inv = 1.f/volume.voxel_size;

    dim3 block(32, 8);
    dim3 grid (divUp (points.cols(), block.x), divUp (points.rows(), block.y));

    raycast_kernel<<<grid, block>>>(rc, (PtrStepSz<Point>)points, normals);
    cudaSafeCall (hipGetLastError ());
}

////////////////////////////////////////////////////////////////////////////////////////
/// Volume cloud exctraction

namespace kfusion
{
    namespace device
    {
        ////////////////////////////////////////////////////////////////////////////////////////
        ///// Prefix Scan utility

        enum ScanKind { exclusive, inclusive };

        template<ScanKind Kind, class T>
        __kf_device__ T scan_warp ( volatile T *ptr, const unsigned int idx = threadIdx.x )
        {
            const unsigned int lane = idx & 31;       // index of thread in warp (0..31)

            if (lane >=  1) ptr[idx] = ptr[idx -  1] + ptr[idx];
            if (lane >=  2) ptr[idx] = ptr[idx -  2] + ptr[idx];
            if (lane >=  4) ptr[idx] = ptr[idx -  4] + ptr[idx];
            if (lane >=  8) ptr[idx] = ptr[idx -  8] + ptr[idx];
            if (lane >= 16) ptr[idx] = ptr[idx - 16] + ptr[idx];

            if (Kind == inclusive)
                return ptr[idx];
            else
                return (lane > 0) ? ptr[idx - 1] : 0;
        }


        __device__ int global_count_vc = 0;
        __device__ int output_count_vc;
        __device__ unsigned int blocks_done_vc = 0;


        struct FullScan6
        {
            enum
            {
                CTA_SIZE_X = 32,
                CTA_SIZE_Y = 6,
                CTA_SIZE = CTA_SIZE_X * CTA_SIZE_Y,

                MAX_LOCAL_POINTS = 3
            };

            TsdfVolume volume;
            Aff3f aff;

            FullScan6(const TsdfVolume& vol) : volume(vol) {}

            __kf_device__ float fetch(int x, int y, int z, float& weight) const
            {
                return unpack_tsdf(*volume(x, y, z), weight);
            }

            __kf_device__ void operator () (PtrSz<Point> output) const
            {
                int x = threadIdx.x + blockIdx.x * CTA_SIZE_X;
                int y = threadIdx.y + blockIdx.y * CTA_SIZE_Y;
#if __CUDA_ARCH__ < 200
                __shared__ int cta_buffer[CTA_SIZE];
#endif

#if __CUDA_ARCH__ >= 120
                if (__all_sync (FULL_MASK, x >= volume.dims.x) || __all_sync (FULL_MASK, y >= volume.dims.y))
                    return;
#else
                if (Emulation::All(x >= volume.dims.x, cta_buffer) || Emulation::All(y >= volume.dims.y, cta_buffer))
                    return;
#endif

                float3 V;
                V.x = (x + 0.5f) * volume.voxel_size.x;
                V.y = (y + 0.5f) * volume.voxel_size.y;

                int ftid = Block::flattenedThreadId ();

                for (int z = 0; z < volume.dims.z - 1; ++z)
                {
                    float3 points[MAX_LOCAL_POINTS];
                    int local_count = 0;

                    if (x < volume.dims.x && y < volume.dims.y)
                    {
                        float W;
                        float F = fetch(x, y, z, W);

                        if (W >= MIN_TSDF_WEIGHT && F != 1.f)
                        {
                            V.z = (z + 0.5f) * volume.voxel_size.z;

                            //process dx
                            if (x + 1 < volume.dims.x)
                            {
                                float Wn;
                                float Fn = fetch(x + 1, y, z, Wn);

                                if (Wn >= MIN_TSDF_WEIGHT && Fn != 1.f)
                                    if ((F > 0 && Fn < 0) || (F < 0 && Fn > 0))
                                    {
                                        float3 p;
                                        p.y = V.y;
                                        p.z = V.z;

                                        float Vnx = V.x + volume.voxel_size.x;

                                        float d_inv = 1.f / (fabs (F) + fabs (Fn));
                                        p.x = (V.x * fabs (Fn) + Vnx * fabs (F)) * d_inv;

                                        points[local_count++] = aff * p;
                                    }
                            }  /* if (x + 1 < volume.dims.x) */

                            //process dy
                            if (y + 1 < volume.dims.y)
                            {
                                float Wn;
                                float Fn = fetch (x, y + 1, z, Wn);

                                if (Wn >= MIN_TSDF_WEIGHT && Fn != 1.f)
                                    if ((F > 0 && Fn < 0) || (F < 0 && Fn > 0))
                                    {
                                        float3 p;
                                        p.x = V.x;
                                        p.z = V.z;

                                        float Vny = V.y + volume.voxel_size.y;

                                        float d_inv = 1.f / (fabs (F) + fabs (Fn));
                                        p.y = (V.y * fabs (Fn) + Vny * fabs (F)) * d_inv;

                                        points[local_count++] = aff * p;
                                    }
                            } /*  if (y + 1 < volume.dims.y) */

                            //process dz
                            //if (z + 1 < volume.dims.z) // guaranteed by loop
                            {
                                float Wn;
                                float Fn = fetch (x, y, z + 1, Wn);

                                if (Wn >= MIN_TSDF_WEIGHT && Fn != 1.f)
                                    if ((F > 0 && Fn < 0) || (F < 0 && Fn > 0))
                                    {
                                        float3 p;
                                        p.x = V.x;
                                        p.y = V.y;

                                        float Vnz = V.z + volume.voxel_size.z;

                                        float d_inv = 1.f / (fabs (F) + fabs (Fn));
                                        p.z = (V.z * fabs (Fn) + Vnz * fabs (F)) * d_inv;

                                        points[local_count++] = aff * p;
                                    }
                            } /* if (z + 1 < volume.dims.z) */
                        } /* if (W != 0 && F != 1.f) */
                    } /* if (x < volume.dims.x && y < volume.dims.y) */

#if __CUDA_ARCH__ >= 200
                    ///not we fulfilled points array at current iteration
                    int total_warp = __popc (__ballot_sync (FULL_MASK, local_count > 0)) + __popc (__ballot_sync (FULL_MASK, local_count > 1)) + __popc (__ballot_sync (FULL_MASK, local_count > 2));
#else
                    int tid = Block::flattenedThreadId();
                    cta_buffer[tid] = local_count;
                    int total_warp = Emulation::warp_reduce(cta_buffer, tid);
#endif
                    __shared__ float storage_X[CTA_SIZE * MAX_LOCAL_POINTS];
                    __shared__ float storage_Y[CTA_SIZE * MAX_LOCAL_POINTS];
                    __shared__ float storage_Z[CTA_SIZE * MAX_LOCAL_POINTS];

                    if (total_warp > 0)
                    {
                        int lane = Warp::laneId ();
                        int storage_index = (ftid >> Warp::LOG_WARP_SIZE) * Warp::WARP_SIZE * MAX_LOCAL_POINTS;

                        volatile int* cta_buffer = (int*)(storage_X + storage_index);

                        cta_buffer[lane] = local_count;
                        int offset = scan_warp<exclusive>(cta_buffer, lane);

                        if (lane == 0)
                        {
                            int old_global_count_vc = atomicAdd (&global_count_vc, total_warp);
                            cta_buffer[0] = old_global_count_vc;
                        }
                        int old_global_count_vc = cta_buffer[0];

                        for (int l = 0; l < local_count; ++l)
                        {
                            storage_X[storage_index + offset + l] = points[l].x;
                            storage_Y[storage_index + offset + l] = points[l].y;
                            storage_Z[storage_index + offset + l] = points[l].z;
                        }

                        Point *pos = output.data + old_global_count_vc + lane;
                        for (int idx = lane; idx < total_warp; idx += Warp::STRIDE, pos += Warp::STRIDE)
                        {
                            float x = storage_X[storage_index + idx];
                            float y = storage_Y[storage_index + idx];
                            float z = storage_Z[storage_index + idx];
                            *pos = make_float4(x, y, z, 0.f);
                        }

                        bool full = (old_global_count_vc + total_warp) >= output.size;

                        if (full)
                            break;
                    }

                } /* for(int z = 0; z < volume.dims.z - 1; ++z) */


                ///////////////////////////
                // prepare for future scans
                if (ftid == 0)
                {
                    unsigned int total_blocks = gridDim.x * gridDim.y * gridDim.z;
                    unsigned int value = atomicInc (&blocks_done_vc, total_blocks);

                    //last block
                    if (value == total_blocks - 1)
                    {
                        output_count_vc = min ((int)output.size, global_count_vc);
                        blocks_done_vc = 0;
                        global_count_vc = 0;
                    }
                }
            }
        };



        __global__ void extract_kernel(const FullScan6 fs, PtrSz<Point> output) { fs(output); }



        struct ExtractNormals
        {
            typedef float8 float8;

            TsdfVolume volume;
            PtrSz<Point> points;
            float3 voxel_size_inv;
            float3 gradient_delta;
            Aff3f aff;
            Mat3f Rinv;

            ExtractNormals(const TsdfVolume& vol) : volume(vol)
            {
                voxel_size_inv.x = 1.f/volume.voxel_size.x;
                voxel_size_inv.y = 1.f/volume.voxel_size.y;
                voxel_size_inv.z = 1.f/volume.voxel_size.z;
            }

            __kf_device__ int3 getVoxel (const float3& p) const
            {
                //rounding to nearest even
                int x = __float2int_rn (p.x * voxel_size_inv.x);
                int y = __float2int_rn (p.y * voxel_size_inv.y);
                int z = __float2int_rn (p.z * voxel_size_inv.z);
                return make_int3 (x, y, z);
            }

            __kf_device__ void operator () (float4* output) const
            {
                int idx = threadIdx.x + blockIdx.x * blockDim.x;

                if (idx >= points.size)
                    return;

                const float qnan = numeric_limits<float>::quiet_NaN ();
                float3 n = make_float3 (qnan, qnan, qnan);

                float3 point = Rinv * (tr(points.data[idx]) - aff.t);
                int3 g = getVoxel (point);

                if (g.x > 1 && g.y > 1 && g.z > 1 && g.x < volume.dims.x - 2 && g.y < volume.dims.y - 2 && g.z < volume.dims.z - 2)
                {
                    float3 t;

                    t = point;
                    t.x += gradient_delta.x;;
                    float Fx1 = interpolate(volume, t * voxel_size_inv);

                    t = point;
                    t.x -= gradient_delta.x;
                    float Fx2 = interpolate(volume, t * voxel_size_inv);

                    n.x = __fdividef(Fx1 - Fx2, gradient_delta.x);

                    t = point;
                    t.y += gradient_delta.y;
                    float Fy1 = interpolate(volume, t * voxel_size_inv);

                    t = point;
                    t.y -= gradient_delta.y;
                    float Fy2 = interpolate(volume, t * voxel_size_inv);

                    n.y = __fdividef(Fy1 - Fy2, gradient_delta.y);

                    t = point;
                    t.z += gradient_delta.z;
                    float Fz1 = interpolate(volume, t * voxel_size_inv);

                    t = point;
                    t.z -= gradient_delta.z;
                    float Fz2 = interpolate(volume, t * voxel_size_inv);

                    n.z = __fdividef(Fz1 - Fz2, gradient_delta.z);

                    n = normalized (aff.R * n);
                }

                output[idx] = make_float4(n.x, n.y, n.z, 0);
            }
        };

        __global__ void extract_normals_kernel (const ExtractNormals en, float4* output) { en(output); }
    }
}

size_t kfusion::device::extractCloud (const TsdfVolume& volume, const Aff3f& aff, PtrSz<Point> output)
{
    typedef FullScan6 FS;
    FS fs(volume);
    fs.aff = aff;

    dim3 block (FS::CTA_SIZE_X, FS::CTA_SIZE_Y);
    dim3 grid (divUp (volume.dims.x, block.x), divUp (volume.dims.y, block.y));

    extract_kernel<<<grid, block>>>(fs, output);
    cudaSafeCall ( hipGetLastError () );
    cudaSafeCall (hipDeviceSynchronize ());

    int size;
    cudaSafeCall ( hipMemcpyFromSymbol(&size, HIP_SYMBOL(output_count_vc), sizeof(size)) );
    return (size_t)size;
}

void kfusion::device::extractNormals (const TsdfVolume& volume, const PtrSz<Point>& points, const Aff3f& aff, const Mat3f& Rinv, float gradient_delta_factor, float4* output)
{
    ExtractNormals en(volume);
    en.points = points;
    en.gradient_delta = volume.voxel_size * gradient_delta_factor;
    en.aff = aff;
    en.Rinv = Rinv;

    dim3 block (256);
    dim3 grid (divUp ((int)points.size, block.x));

    extract_normals_kernel<<<grid, block>>>(en, output);
    cudaSafeCall ( hipGetLastError () );
    cudaSafeCall (hipDeviceSynchronize ());
}
